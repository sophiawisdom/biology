#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#define ITER 10000000
#define NUM_BLOCKS 32
#define THREADS_PER_BLOCK 128
#define TOTAL_THREADS (NUM_BLOCKS * THREADS_PER_BLOCK)
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#include <device.cu> // setup and generate kernel.

// Benchmark depends on size, but ranges from ~1600 million matings/s (@100k organisms) to
// 26000 million matings/s (@16m organisms). Speed increases the more organisms there are primarily
// And the more generations it's running for secondarily.

void write_bytearray(void *array, int length, char* filename){
  int fd = open(filename,O_WRONLY | O_CREAT);
  int result = write(fd,array,length);
  close(fd);
  return;
}

void print_threadinfo(thread_info tinfo){
  printf("num_sims: %d\nthresh_aa: %d\nthresh_ab: %d\nthresh_bb: %d\nscaledown_factor: %d\n",
    tinfo.num_sims,tinfo.thresh_aa,tinfo.thresh_ab,tinfo.thresh_bb,tinfo.scaledown_factor);
}

long long * get_seeds(int num_seeds){
  long long *host_seeds = (long long *) malloc(num_seeds*sizeof(long long));
  int fd = open("/dev/urandom",O_RDONLY);
  read(fd,host_seeds,num_seeds*sizeof(long long));
  close(fd);
  return host_seeds;
}

int main(int argc, char **argv){

  #ifdef DEBUG
  int num_generations = 10;
 #else
  int num_generations = 10000;
  if (argc == 3){
  	num_generations = atoi(argv[1]);
  }
  #endif

  int num_organisms = 1024*1024;
  if (argc == 3) {
	  num_organisms = atoi(argv[2]);
  }

  if (__builtin_popcount(num_organisms) != 1){
  	printf("num_organisms must be a power of 2\n");
  	exit(EXIT_FAILURE);
  }

  printf("Running simulation for %d generations with %d organisms for %d total_threads\n",num_generations, num_organisms, TOTAL_THREADS);

  long long * host_seeds = get_seeds(TOTAL_THREADS);
  long long * device_seeds;
  CUDA_CALL(hipMalloc(&device_seeds,TOTAL_THREADS*sizeof(long long)));
  CUDA_CALL(hipMemcpy(device_seeds, host_seeds, TOTAL_THREADS*sizeof(long long), hipMemcpyHostToDevice));

  int * host_results = (int *) malloc(TOTAL_THREADS*sizeof(int));
  int * device_results;
  CUDA_CALL(hipMalloc(&device_results, TOTAL_THREADS*sizeof(int)));
  CUDA_CALL(hipMemset(device_results, 0, TOTAL_THREADS*sizeof(int)));

  hiprandState *device_rand_state;
  CUDA_CALL(hipMalloc(&device_rand_state,TOTAL_THREADS*sizeof(hiprandState)));

  setup_kernel<<<NUM_BLOCKS,THREADS_PER_BLOCK>>> (device_rand_state, device_seeds);

  thread_info t_info;
  t_info.num_sims = (num_organisms/TOTAL_THREADS);
  t_info.thresh_aa = num_organisms>>2;
  t_info.thresh_ab = (num_organisms>>2) * 3;
  t_info.thresh_bb = num_organisms;
  int scaledown = 0;
  for (int i = 32; i > 0; i--){
  	if (num_organisms == (1 << i)){
  		scaledown = (32-i);
  		break;
  	}
  }
  t_info.scaledown_factor = scaledown;
  t_info.results = device_results;

  int *overall_results = (int *) malloc(num_generations*3*sizeof(int));

  struct timeval start;
  struct timeval end;
  gettimeofday(&start,NULL);
  unsigned long long start_time = (start.tv_sec*1000000) + start.tv_usec;
  unsigned long long last_printed = start_time;


  for (int generation = 0; generation < num_generations; generation++){

    #ifdef DEBUG
    printf("Before running kernel for generation %d, tinfo looked like:\n",generation);
    print_threadinfo(t_info);
    #endif

    generate_kernel<<<NUM_BLOCKS,THREADS_PER_BLOCK>>> (device_rand_state, t_info);

    int num_aa = 0;
    int num_ab = 0;
    int num_bb = 0;

    CUDA_CALL(hipMemcpy(host_results, device_results, TOTAL_THREADS*sizeof(int), hipMemcpyDeviceToHost));
    // We don't have to worry about cpu/gpu synchronization because hipMemcpy will run after generate_kernel

    for (int i = 0; i < TOTAL_THREADS; i++){
      int k = host_results[i];

      short num_aa_short = k >> 16; // upper 16 bits
      short num_ab_short = k & 65535; // lower 16 bits
      // More efficient methods of extraction that explicitly extract 16 bits in asm, but not super
      // important
      short num_bb_short = (t_info.num_sims - (num_aa_short+num_ab_short));

      #ifdef DEBUG
      if (i % 300 == 299) {
        printf("k = %08x, num_aa_short = %d, num_ab_short = %d, num_bb_short = %d, generation %d\n",k,num_aa_short,num_ab_short,num_bb_short,generation);
      }
      #endif

      num_aa += num_aa_short;
      num_ab += num_ab_short;
      num_bb += num_bb_short;
    }


    t_info.thresh_aa = num_aa;
    t_info.thresh_ab = num_aa + num_ab;
    int index = generation*3;
    overall_results[index] = num_aa;
    overall_results[index+1] = num_ab;
    overall_results[index+2] = num_bb;

    if (generation % 10000 == 0 && generation != 0){
      gettimeofday(&end, NULL);
      unsigned long long end_time = end.tv_sec*1000000 + end.tv_usec;

      if (end_time - last_printed >= (30*1000*1000)){
      	float since_start = (end_time-start_time)/1000000.0;
      	float since_printed = (end_time-last_printed)/1000000.0;
      	float ratio_done = generation/num_generations;
      	float time_left = since_printed/ratio_done;
      	printf("Currently on generation %d. It's been %f seconds. Projected to be %f seconds until done.\n", generation, since_start, since_printed, time_left);
      	last_printed = end_time;
      }
    }
  }

  int i = (num_generations-5)*3;

  printf("At end, num_aa = %d, num_ab = %d, num_bb = %d\n", overall_results[i], overall_results[i+1], overall_results[i+2]);

  gettimeofday(&end, NULL);
  unsigned long long end_time = (end.tv_sec*1000000) + end.tv_usec;

  float time_taken = (end_time-start_time)/1000000.0;

  unsigned long long matings = num_generations;
  matings = matings << (32-t_info.scaledown_factor);

  printf("Took %f seconds %llu million matings/s\n", time_taken, matings/(end_time-start_time));

  #ifdef DEBUG // 40694899
  printf("Freeing host memory\n");
  #endif
  free(host_seeds);
  free(host_results);
  free(overall_results);
  #ifdef DEBUG
  printf("Freed host memory\n");

  printf("Attempting to free device_results\n");
  #endif
  CUDA_CALL(hipFree(device_results));
  #ifdef DEBUG
  printf("Freed device results\n");
  #endif
  CUDA_CALL(hipFree(device_rand_state));
  #ifdef DEBUG
  printf("Freed device_rand_state\n");
  #endif
  CUDA_CALL(hipFree(device_seeds));
  #ifdef DEBUG
  printf("Freed device_seeds\n");
  #endif
  return 0;
}
