#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdio.h>
#include <assert.h>
#define ITER 10000000
#define NUM_BLOCKS 32
#define THREADS_PER_BLOCK 32
#define TOTAL_THREADS NUM_BLOCKS * THREADS_PER_BLOCK
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

struct thread_info {
  int num_sims; // How many simulations a thread should be doing

  int thresh_aa;
  int thresh_ab;
  int thresh_bb; // implied to equal num_sims and be above 65536

  /* 0 -> thresh_aa -> thresh_ab -> thresh_bb
   *   aa           ab          bb
   * thresh_aa is aa, thresh_ab is ab, thresh_bb is bb, or it should be. I don't think this is yet
   * the case.
  */

  int scaledown_factor; // equal to 32 - (log(thresh_bb,2)). How much int should be scaled down

  int * results; // [num_aa, num_ab] with num_bb being implicitly total - (num_aa+num_ab)
  // Makes more sense in terms of limiting the number of memory accesses. Turns two memory acceses into 2
}
// Does it matter whether I pass a struct with arguments or series of arguments?

__global__ void setup_kernel(hiprandState *state, long long *seeds){

  int idx = threadIdx.x+blockIdx.x*THREADS_PER_BLOCK;
  long long seed = 1234;//seeds[idx];
  hiprand_init(seed, idx, 0, &state[idx]); // &state[idx] != state+idx 
}

__global__ void generate_kernel(hiprandState *my_curandstate, thread_info t_info){
  int idx = threadIdx.x+blockIdx.x*THREADS_PER_BLOCK;
  short num_aa = 0;
  short num_ab = 0;
  short num_bb = 0;
  hiprandState localCurandState = my_curandstate[idx];
  threadinfo local_tinfo = t_info; // IDK if this is good practice; the idea is to load the thread info locally instead of getting it from the main source

  int num_one_parent = 0;
  for (int i = 0; i < (local_tinfo.num_sims>>5); i++){
    num_one_parent += __popc(hiprand(&localCurandState)); // 64 bit version would be somewhat better, but whatevs
  }

  int num_two_parents = local_tinfo.num_sims-num_one_parent;

  for (int i = 0; i < num_one_parent; i++){
    unsigned int rand_num = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

    if (rand_num < local_tinfo.thresh_aa){
      num_aa += 1;
    }

    else if (rand_num < local_tinfo.thresh_ab){
      num_ab += 1;
    }

  }

  for (int i = 0; i < num_two_parents; i++){
    unsigned int p1_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;
    char allele = 0;
    if (p1_index > local_tinfo.thresh_ab){
      unsigned int p2_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

      if (p2_index < local_tinfo.thresh_aa){
        num_ab += 1;
      }

      else if (p2_index < local_tinfo.thresh_ab) {

        if (p1_index&1 == 0){
          num_ab += 1;
        }

      }

    }

    else if (p1_index > local_tinfo.thresh_aa) {
      unsigned int p2_index = hiprand(&localCurandState) >>local_tinfo.scaledown_factor;

      if (p1_index&1){ // first b sequence

        if (p2_index < local_tinfo.thresh_aa){
          num_ab += 1;
        }

        if (p2_index < local_tinfo.thresh_ab){

          if (p2_index&0) {
            num_ab += 1;
          }

        }

      }

      else { // first a sequence

        if (p2_index > local_tinfo.thresh_ab){
          num_ab += 1;
        }

        else if (p2_index > local_tinfo.thresh_aa){

          if (p2_index&1) {
            num_ab += 1;
          }

          else {
            num_aa += 1;
          }
        }

        else {
          num_aa += 1;
        }
      }
    }

    else {
      unsigned int p2_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

      if (p2_index > local_tinfo.thresh_ab){
        num_ab += 1;
      }

      else if (p2_index > local_tinfo.thresh_aa){

        if (p2_index&1){
          num_ab += 1;
        }

        else {
          num_aa += 1;
        }

      }

      else {
        num_aa += 1;
      }

    }
  }

  int result = num_aa << 16 + num_ab; // Try to make sure the compiler knows this is an int
  thread_info.results[idx] = result;
}

int main(){
  
}


int main(){
  printf("Total number of threads: %d. Allocating rand state memory\n",TOTAL_THREADS);

  hiprandState *d_state;
  CUDA_CALL(hipMalloc(&d_state, sizeof(hiprandState)));
  printf("Allocated random state memory, reading host seeds\n")

  long long *host_seeds = (long long *) malloc(TOTAL_THREADS*sizeof(long long));
  int fd = open("/dev/urandom",O_RDONLY);
  read(fd,host_seeds,TOTAL_THREADS*sizeof(long long));
  close(fd);
  printf("Finished reading host seeds, copying to device\n");

  long long *device_seeds;
  CUDA_CALL(hipMalloc(&device_seeds, TOTAL_THREADS*sizeof(long long)));
  CUDA_CALL(hipMemcpy(device_seeds, host_seeds, TOTAL_THREADS*sizeof(long long), hipMemcpyHostToDevice));
  printf("Running setup kernel\n");

  setup_kernel<<<NUM_BLOCKS,THREADS_PER_BLOCK>>>(d_state, device_seeds);
  printf("Finished setup_kernel, setting up device_results on GPU\n");

  unsigned int *device_results;
  CUDA_CALL(hipMalloc(&device_results, TOTAL_THREADS*sizeof(int)));
  CUDA_CALLcudaMemset(device_results, 0, TOTAL_THREADS*sizeof(int));
  unsigned int *host_results = (unsigned int *)malloc(TOTAL_THREADS*sizeof(int));
  printf("Finished device and host results setup, sending kernel now\n");

  generate_kernel<<<NUM_BLOCKS,THREADS_PER_BLOCK>>>(d_state, device_results);
  printf("Finished sending kernel, synchronizing now.\n");

  hipDeviceSynchronize();
  printf("Finished synchronizing, tallying now.\n");

  hipMemcpy(host_results, device_results, TOTAL_THREADS*sizeof(int), hipMemcpyDeviceToHost);

  unsigned long long sum = 0;
  for (int i = 0; i < TOTAL_THREADS; i++){
    sum += host_results[i];
  }
  printf("Sum of %d threads random numbers is %llu. Average is %llu\n",TOTAL_THREADS,sum,sum>>10);

  hipFree(&device_seeds);
  hipFree(&d_state);
}
