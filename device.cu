#include "hip/hip_runtime.h"
struct thread_info {
  int num_sims; // How many simulations a thread should be doing

  int thresh_aa;
  int thresh_ab;
  int thresh_bb; // implied to equal num_sims and be above 65536

  /* 0 -> thresh_aa -> thresh_ab -> thresh_bb
   *   aa           ab          bb
   * thresh_aa is aa, thresh_ab is ab, thresh_bb is bb, or it should be. I don't think this is yet
   * the case.
  */

  int scaledown_factor; // equal to 32 - (log(thresh_bb,2)). How much int should be scaled down

  int * results; // [num_aa, num_ab] with num_bb being implicitly total - (num_aa+num_ab)
  // Makes more sense in terms of limiting the number of memory accesses. Turns two memory acceses into 2
};
// Does it matter whether I pass a struct with arguments or series of arguments?

__global__ void setup_kernel(hiprandState *state, long long *seeds) {
  int idx = threadIdx.x+blockIdx.x*THREADS_PER_BLOCK;
  long long seed = seeds[idx];
  hiprand_init(seed, idx, 0, &state[idx]); // &state[idx] != state+idx 
}

__global__ void generate_kernel(hiprandState *curandstate, thread_info t_info){
  int idx = threadIdx.x+blockIdx.x*THREADS_PER_BLOCK;
  hiprandState localCurandState = curandstate[idx];

  unsigned short num_aa = 0;
  unsigned short num_ab = 0;
  // num_bb is implicit and equal to (tinfo.num_sims-(num_aa+num_ab))
  thread_info local_tinfo = t_info;
  // IDK if this is good practice; the idea is to load the thread info locally instead of getting it from the main source

  int num_one_parent = 0;
  for (int i = 0; i < (local_tinfo.num_sims>>5); i++){
  	int rand_int = hiprand(&localCurandState);
    num_one_parent += __popc(rand_int); // 64 bit version would be somewhat better, but whatevs
  }

  int num_two_parents = local_tinfo.num_sims - num_one_parent;

//  t_info.results[idx] = num_one_parent;
//  curandstate[idx] = localCurandState;

//  local_tinfo.results[idx] = hiprand(&localCurandState) >> 16;
//  return;

  for (int i = 0; i < num_one_parent; i++) {
    unsigned int rand_num = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

    if (rand_num < local_tinfo.thresh_aa){
      num_aa++;
    }

    else if (rand_num < local_tinfo.thresh_ab){
      num_ab++;
    }

    // Implict else num_bb++;

  }

  for (int i = 0; i < num_two_parents; i++){
    unsigned int p1_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

    if (p1_index > local_tinfo.thresh_ab) {
    	// First bit b
      unsigned int p2_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

      if (p2_index < local_tinfo.thresh_aa) {
        num_ab++;
      }

      else if (p2_index < local_tinfo.thresh_ab) {

        if ((p2_index&1) == 0) {
          num_ab++;
        }

        // else {num_bb++}
      }

      // else {num_bb++}

    }

    else if (p1_index > local_tinfo.thresh_aa) {
      unsigned int p2_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

      if ((p1_index&1) == 1){ // First bit B, same as above.

        if (p2_index < local_tinfo.thresh_aa){
          num_ab++;
        }

        else if (p2_index < local_tinfo.thresh_ab){

          if ((p2_index&1) == 0) { // Second bit 0
            num_ab++;
          }

          // num_bb++
        }

        // num_bb++

      }

      else { // First bit a

        if (p2_index > local_tinfo.thresh_ab) { // Second bit B
          num_ab++;
        }

        else if (p2_index > local_tinfo.thresh_aa){ // second bit AB

          if ((p2_index&1) == 1) { // Second bit B
            num_ab++;
          }

          else { // Second bit A
            num_aa++;
          }

        }

        else { // Second bit A
          num_aa++;
        }
      }
    }

    else { // First bit A
      unsigned int p2_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

      if (p2_index > local_tinfo.thresh_ab){
        num_ab++;
      }

      else if (p2_index > local_tinfo.thresh_aa){

        if ((p2_index&1) == 1){
          num_ab++;
        }

        else {
          num_aa++;
        }

      }

      else {
        num_aa++;
      }

    }
  }

  curandstate[idx] = localCurandState;

  int result = (int)(num_aa);
  result = result << 16;
  result += num_ab; // Try to make sure the compiler knows this is an int
  local_tinfo.results[idx] = result;
}