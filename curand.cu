
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#define ITER 10000000
#define NUM_BLOCKS 32
#define THREADS_PER_BLOCK 128
#define TOTAL_THREADS (NUM_BLOCKS * THREADS_PER_BLOCK)
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

struct thread_info {
  int num_sims; // How many simulations a thread should be doing

  int thresh_aa;
  int thresh_ab;
  int thresh_bb; // implied to equal num_sims and be above 65536

  /* 0 -> thresh_aa -> thresh_ab -> thresh_bb
   *   aa           ab          bb
   * thresh_aa is aa, thresh_ab is ab, thresh_bb is bb, or it should be. I don't think this is yet
   * the case.
  */

  int scaledown_factor; // equal to 32 - (log(thresh_bb,2)). How much int should be scaled down

  int * results; // [num_aa, num_ab] with num_bb being implicitly total - (num_aa+num_ab)
  // Makes more sense in terms of limiting the number of memory accesses. Turns two memory acceses into 2
};
// Does it matter whether I pass a struct with arguments or series of arguments?

__global__ void setup_kernel(hiprandState *state, long long *seeds) {
  int idx = threadIdx.x+blockIdx.x*THREADS_PER_BLOCK;
  long long seed = seeds[idx];
  hiprand_init(seed, idx, 0, &state[idx]); // &state[idx] != state+idx 
}

__global__ void generate_kernel(hiprandState *curandstate, thread_info t_info){
  int idx = threadIdx.x+blockIdx.x*THREADS_PER_BLOCK;
  hiprandState localCurandState = curandstate[idx];

  unsigned short num_aa = 0;
  unsigned short num_ab = 0;
  // num_bb is implicit and equal to (tinfo.num_sims-(num_aa+num_ab))
  thread_info local_tinfo = t_info;
  // IDK if this is good practice; the idea is to load the thread info locally instead of getting it from the main source

  int num_one_parent = 0;
  for (int i = 0; i < (local_tinfo.num_sims>>5); i++){
  	int rand_int = hiprand(&localCurandState);
    num_one_parent += __popc(rand_int); // 64 bit version would be somewhat better, but whatevs
  }

  int num_two_parents = local_tinfo.num_sims - num_one_parent;

//  t_info.results[idx] = num_one_parent;
//  curandstate[idx] = localCurandState;

//  local_tinfo.results[idx] = curand(&localCurandState) >> 16;
//  return;

  for (int i = 0; i < num_one_parent; i++) {
    unsigned int rand_num = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

    if (rand_num < local_tinfo.thresh_aa){
      num_aa++;
    }

    else if (rand_num < local_tinfo.thresh_ab){
      num_ab++;
    }

    // Implict else num_bb++;

  }

  for (int i = 0; i < num_two_parents; i++){
    unsigned int p1_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

    if (p1_index > local_tinfo.thresh_ab) {
    	// First bit b
      unsigned int p2_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

      if (p2_index < local_tinfo.thresh_aa) {
        num_ab++;
      }

      else if (p2_index < local_tinfo.thresh_ab) {

        if ((p2_index&1) == 0) {
          num_ab++;
        }

        // else {num_bb++}
      }

      // else {num_bb++}

    }

    else if (p1_index > local_tinfo.thresh_aa) {
      unsigned int p2_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

      if ((p1_index&1) == 1){ // First bit B, same as above.

        if (p2_index < local_tinfo.thresh_aa){
          num_ab++;
        }

        else if (p2_index < local_tinfo.thresh_ab){

          if ((p2_index&1) == 0) { // Second bit 0
            num_ab++;
          }

          // num_bb++
        }

        // num_bb++

      }

      else { // First bit a

        if (p2_index > local_tinfo.thresh_ab) { // Second bit B
          num_ab++;
        }

        else if (p2_index > local_tinfo.thresh_aa){ // second bit AB

          if ((p2_index&1) == 1) { // Second bit B
            num_ab++;
          }

          else { // Second bit A
            num_aa++;
          }

        }

        else { // Second bit A
          num_aa++;
        }
      }
    }

    else { // First bit A
      unsigned int p2_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

      if (p2_index > local_tinfo.thresh_ab){
        num_ab++;
      }

      else if (p2_index > local_tinfo.thresh_aa){

        if ((p2_index&1) == 1){
          num_ab++;
        }

        else {
          num_aa++;
        }

      }

      else {
        num_aa++;
      }

    }
  }

  curandstate[idx] = localCurandState;

  int result = (int)(num_aa);
  result = result << 16;
  result += num_ab; // Try to make sure the compiler knows this is an int
  local_tinfo.results[idx] = result;
}

void write_bytearray(void *array, int length, char* filename){
  int fd = open(filename,O_WRONLY | O_CREAT);
  int result = write(fd,array,length);
  close(fd);
  return;
}

void print_threadinfo(thread_info tinfo){
  printf("num_sims: %d\nthresh_aa: %d\nthresh_ab: %d\nthresh_bb: %d\nscaledown_factor: %d\n",
    tinfo.num_sims,tinfo.thresh_aa,tinfo.thresh_ab,tinfo.thresh_bb,tinfo.scaledown_factor);
}

long long * get_seeds(int num_seeds){
  long long *host_seeds = (long long *) malloc(num_seeds*sizeof(long long));
  int fd = open("/dev/urandom",O_RDONLY);
  read(fd,host_seeds,num_seeds*sizeof(long long));
  close(fd);
  return host_seeds;
}

int main(int argc, char **argv){

  #ifdef DEBUG
  int num_generations = 10;
 #else
  int num_generations = 10000;
  if (argc == 3){
  	num_generations = atoi(argv[1]);
  }
  #endif

  int num_organisms = 1024*1024;
  if (argc == 3) {
	  num_organisms = atoi(argv[2]);
  }

  if (__builtin_popcount(num_organisms) != 1){
  	printf("num_organisms must be a power of 2\n");
  	exit(EXIT_FAILURE);
  }

  printf("Running simulation for %d generations with %d organisms for %d total_threads\n",num_generations, num_organisms, TOTAL_THREADS);

  long long * host_seeds = get_seeds(TOTAL_THREADS);
  long long * device_seeds;
  CUDA_CALL(hipMalloc(&device_seeds,TOTAL_THREADS*sizeof(long long)));
  CUDA_CALL(hipMemcpy(device_seeds, host_seeds, TOTAL_THREADS*sizeof(long long), hipMemcpyHostToDevice));

  int * host_results = (int *) malloc(TOTAL_THREADS*sizeof(int));
  int * device_results;
  CUDA_CALL(hipMalloc(&device_results, TOTAL_THREADS*sizeof(int)));
  CUDA_CALL(hipMemset(device_results, 0, TOTAL_THREADS*sizeof(int)));

  hiprandState *device_rand_state;
  CUDA_CALL(hipMalloc(&device_rand_state,TOTAL_THREADS*sizeof(hiprandState)));

  setup_kernel<<<NUM_BLOCKS,THREADS_PER_BLOCK>>> (device_rand_state, device_seeds);

  thread_info t_info;
  t_info.num_sims = (num_organisms/TOTAL_THREADS);
  t_info.thresh_aa = num_organisms>>2;
  t_info.thresh_ab = (num_organisms>>2) * 3;
  t_info.thresh_bb = num_organisms;
  int scaledown = 0;
  for (int i = 32; i > 0; i--){
  	if (num_organisms == (1 << i)){
  		scaledown = (32-i);
  		break;
  	}
  }
  t_info.scaledown_factor = scaledown;
  t_info.results = device_results;

  int *overall_results = (int *) malloc(num_generations*3*sizeof(int));

  struct timeval start;
  struct timeval end;
  gettimeofday(&start,NULL);
  unsigned long long start_time = (start.tv_sec*1000000) + start.tv_usec;
  unsigned long long last_printed = start_time;


  for (int generation = 0; generation < num_generations; generation++){

    #ifdef DEBUG
    printf("Before running kernel for generation %d, tinfo looked like:\n",generation);
    print_threadinfo(t_info);
    #endif

    generate_kernel<<<NUM_BLOCKS,THREADS_PER_BLOCK>>> (device_rand_state, t_info);

    int num_aa = 0;
    int num_ab = 0;
    int num_bb = 0;

    CUDA_CALL(hipMemcpy(host_results, device_results, TOTAL_THREADS*sizeof(int), hipMemcpyDeviceToHost));
    // We don't have to worry about cpu/gpu synchronization because cudaMemcpy will run after generate_kernel

    for (int i = 0; i < TOTAL_THREADS; i++){
      int k = host_results[i];

      short num_aa_short = k >> 16; // upper 16 bits
      short num_ab_short = k & 65535; // lower 16 bits
      // More efficient methods of extraction that explicitly extract 16 bits in asm, but not super
      // important
      short num_bb_short = (t_info.num_sims - (num_aa_short+num_ab_short));

      #ifdef DEBUG
      if (i % 300 == 299) {
        printf("k = %08x, num_aa_short = %d, num_ab_short = %d, num_bb_short = %d, generation %d\n",k,num_aa_short,num_ab_short,num_bb_short,generation);
      }
      #endif

      num_aa += num_aa_short;
      num_ab += num_ab_short;
      num_bb += num_bb_short;
    }


    t_info.thresh_aa = num_aa;
    t_info.thresh_ab = num_aa + num_ab;
    int index = generation*3;
    overall_results[index] = num_aa;
    overall_results[index+1] = num_ab;
    overall_results[index+2] = num_bb;

    if (generation % 10000 == 0 && generation != 0){
      gettimeofday(&end, NULL);
      unsigned long long end_time = end.tv_sec*1000000 + end.tv_usec;

      if (end_time - last_printed >= (30*1000*1000)){
      	float since_start = (end_time-start_time)/1000000.0;
      	float since_printed = (end_time-last_printed)/1000000.0;
      	float ratio_done = generation/num_generations;
      	float time_left = since_printed/ratio_done;
      	printf("Currently on generation %d. It's been %f seconds. Projected to be %f seconds until done.\n", generation, since_start, since_printed, time_left);
      	last_printed = end_time;
      }
    }
  }

  int i = (num_generations-5)*3;

  printf("At end, num_aa = %d, num_ab = %d, num_bb = %d\n", overall_results[i], overall_results[i+1], overall_results[i+2]);

  gettimeofday(&end, NULL);
  unsigned long long end_time = (end.tv_sec*1000000) + end.tv_usec;

  float time_taken = (end_time-start_time)/1000000.0;

  unsigned long long matings = num_generations;
  matings = matings << (32-t_info.scaledown_factor);

  printf("Took %f seconds %llu million matings/s\n", time_taken, matings/(end_time-start_time));

  #ifdef DEBUG // 40694899
  printf("Freeing host memory\n");
  #endif
  free(host_seeds);
  free(host_results);
  free(overall_results);
  #ifdef DEBUG
  printf("Freed host memory\n");

  printf("Attempting to free device_results\n");
  #endif
  CUDA_CALL(hipFree(device_results));
  #ifdef DEBUG
  printf("Freed device results\n");
  #endif
  CUDA_CALL(hipFree(device_rand_state));
  #ifdef DEBUG
  printf("Freed device_rand_state\n");
  #endif
  CUDA_CALL(hipFree(device_seeds));
  #ifdef DEBUG
  printf("Freed device_seeds\n");
  #endif
  return 0;
}
