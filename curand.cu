
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#define ITER 10000000
#define NUM_BLOCKS 32
#define THREADS_PER_BLOCK 32
#define TOTAL_THREADS (NUM_BLOCKS * THREADS_PER_BLOCK)
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

struct thread_info {
  int num_sims; // How many simulations a thread should be doing

  int thresh_aa;
  int thresh_ab;
  int thresh_bb; // implied to equal num_sims and be above 65536

  /* 0 -> thresh_aa -> thresh_ab -> thresh_bb
   *   aa           ab          bb
   * thresh_aa is aa, thresh_ab is ab, thresh_bb is bb, or it should be. I don't think this is yet
   * the case.
  */

  int scaledown_factor; // equal to 32 - (log(thresh_bb,2)). How much int should be scaled down

  int * results; // [num_aa, num_ab] with num_bb being implicitly total - (num_aa+num_ab)
  // Makes more sense in terms of limiting the number of memory accesses. Turns two memory acceses into 2
};
// Does it matter whether I pass a struct with arguments or series of arguments?

__global__ void setup_kernel(hiprandState *state, long long *seeds){

  int idx = threadIdx.x+blockIdx.x*THREADS_PER_BLOCK;
  long long seed = 1234;//seeds[idx];
  hiprand_init(seed, idx, 0, &state[idx]); // &state[idx] != state+idx 
}

__global__ void generate_kernel(hiprandState *my_curandstate, thread_info t_info){
  int idx = threadIdx.x+blockIdx.x*THREADS_PER_BLOCK;
  unsigned short num_aa = 0;
  unsigned short num_ab = 0;
  hiprandState localCurandState = my_curandstate[idx];
  thread_info local_tinfo = t_info; 
  // IDK if this is good practice; the idea is to load the thread info locally instead of getting it from the main source

  int num_one_parent = 0;
  for (int i = 0; i < (local_tinfo.num_sims>>5); i++){
    num_one_parent += __popc(hiprand(&localCurandState)); // 64 bit version would be somewhat better, but whatevs
  }

  int num_two_parents = local_tinfo.num_sims-num_one_parent;

  for (int i = 0; i < num_one_parent; i++){
    unsigned int rand_num = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

    if (rand_num < local_tinfo.thresh_aa){
      num_aa++;
    }

    else if (rand_num < local_tinfo.thresh_ab){
      num_ab++;
    }

  }

  for (int i = 0; i < num_two_parents; i++){
    unsigned int p1_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;
    if (p1_index > local_tinfo.thresh_ab){
      unsigned int p2_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

      if (p2_index < local_tinfo.thresh_aa){
        num_ab++;
      }

      else if (p2_index < local_tinfo.thresh_ab) {

        if (p1_index&1 == 0){
          num_ab++;
        }

      }

    }

    else if (p1_index > local_tinfo.thresh_aa) {
      unsigned int p2_index = hiprand(&localCurandState) >>local_tinfo.scaledown_factor;

      if (p1_index&1){ // first b sequence

        if (p2_index < local_tinfo.thresh_aa){
          num_ab++;
        }

        if (p2_index < local_tinfo.thresh_ab){

          if (p2_index&0) {
            num_ab++;
          }

        }

      }

      else { // first a sequence

        if (p2_index > local_tinfo.thresh_ab){
          num_ab++;
        }

        else if (p2_index > local_tinfo.thresh_aa){

          if (p2_index&1) {
            num_ab++;
          }

          else {
            num_aa++;
          }
        }

        else {
          num_aa++;
        }
      }
    }

    else {
      unsigned int p2_index = hiprand(&localCurandState) >> local_tinfo.scaledown_factor;

      if (p2_index > local_tinfo.thresh_ab){
        num_ab++;
      }

      else if (p2_index > local_tinfo.thresh_aa){

        if (p2_index&1){
          num_ab++;
        }

        else {
          num_aa++;
        }

      }

      else {
        num_aa++;
      }

    }
  }

  int result = (int)(num_aa);
  result = result << 16;
  result += num_ab; // Try to make sure the compiler knows this is an int
  local_tinfo.results[idx] = result;
}

void write_bytearray(void *array, int length, char* filename){
  int fd = open(filename,O_WRONLY | O_CREAT);
  int result = write(fd,array,length);
  close(fd);
  return;
}

void print_threadinfo(thread_info tinfo){
  printf("num_sims: %d\nthresh_aa: %d\nthresh_ab: %d\nthresh_bb: %d\nscaledown_factor: %d\nresults: %d\n",
    tinfo.num_sims,tinfo.thresh_aa,tinfo.thresh_ab,tinfo.thresh_bb,tinfo.scaledown_factor,tinfo.results);
}

long long * get_seeds(int num_seeds){
  long long *host_seeds = (long long *) malloc(num_seeds*sizeof(long long));
  int fd = open("/dev/urandom",O_RDONLY);
  read(fd,host_seeds,num_seeds*sizeof(long long));
  close(fd);
  return host_seeds;
}

int main(){

  int num_generations = 10000;
  int num_organisms = 1024*1024;

  printf("Running simulation for %d total_threads\n",TOTAL_THREADS);

  long long * host_seeds = get_seeds(TOTAL_THREADS);
  long long * device_seeds;
  CUDA_CALL(hipMalloc(&device_seeds,TOTAL_THREADS*sizeof(long long)));
  CUDA_CALL(hipMemcpy(device_seeds, host_seeds, TOTAL_THREADS*sizeof(long long), hipMemcpyHostToDevice));

  int * host_results = (int *) malloc(TOTAL_THREADS*sizeof(int));
  int * device_results;
  CUDA_CALL(hipMalloc(&device_results, TOTAL_THREADS*sizeof(int)));
  CUDA_CALL(hipMemset(device_results, 0, TOTAL_THREADS*sizeof(int)));

  hiprandState *device_rand_state;
  CUDA_CALL(hipMalloc(&device_rand_state,TOTAL_THREADS*sizeof(hiprandState)));

  setup_kernel<<<NUM_BLOCKS,THREADS_PER_BLOCK>>> (device_rand_state, device_seeds);

  thread_info t_info;
  t_info.num_sims = (num_organisms/TOTAL_THREADS);
  t_info.thresh_aa = 262144;
  t_info.thresh_ab = 786432;
  t_info.thresh_bb = 1048576;
  t_info.scaledown_factor = 12;
  t_info.results = device_results;

  int *overall_results = (int *) malloc(num_generations*3);

  struct timeval start;
  struct timeval end;
  gettimeofday(&start,NULL);
  unsigned long long start_time = start.tv_sec*1000000 + start.tv_usec;

  for (int generation = 0; generation < num_generations; generation++){
    #ifdef DEBUG
    printf("Before running kernel for generation %d, tinfo looked like:\n",generation);
    print_threadinfo(t_info);
    #endif

    generate_kernel<<<NUM_BLOCKS,THREADS_PER_BLOCK>>> (device_rand_state, t_info);

    int num_aa = 0;
    int num_ab = 0;
    int num_bb = 0;

    CUDA_CALL(hipMemcpy(host_results, device_results, TOTAL_THREADS*sizeof(int), hipMemcpyDeviceToHost));
    // We don't have to worry about cpu/gpu synchronization because cudaMemcpy will run after generate_kernel

    for (int i = 0; i < TOTAL_THREADS; i++){
      int k = host_results[i];
      short num_aa_short = k >> 16; // upper 16 bits
      short num_ab_short = k & 65535; // lower 16 bits
      // More efficient methods of extraction that explicitly extract 16 bits in asm, but not super
      // important
      short num_bb_short = (t_info.num_sims - (num_aa_short+num_ab_short));
      #ifdef DEBUG
      if (i % 300 == 299) {
        printf("k = %d, num_aa_short = %d, num_ab_short = %d, generation %d\n",k,num_aa_short,num_ab_short,generation);
      }
      #endif
      num_aa += num_aa_short;
      num_ab += num_ab_short;
      num_bb += num_bb_short;
    }

    #ifdef DEBUG
    if (generation == 0){
      write_bytearray(host_results,TOTAL_THREADS*sizeof(int),"results");
    }
    #endif

    t_info.thresh_aa = num_aa;
    t_info.thresh_ab = num_aa + num_ab;
    int index = generation*3;
    overall_results[index] = num_aa;
    overall_results[index+1] = num_ab;
    overall_results[index+2] = num_bb;
    if (generation % 1000 == 0 && generation != 0){
      gettimeofday(&end, NULL);
      unsigned long long end_time = end.tv_sec*1000000 + end.tv_usec;
      printf("Results for generation %d are %d aa, %d ab and %d bb! Took %.3f seconds.\n",
        generation,num_aa,num_ab,num_bb,(end_time-start_time)/1000000.0);
    }
  }

  printf("Freeing host memory\n");
  free(host_seeds);
  free(host_results);
  free(overall_results);
  printf("Freed host memory\n");

  printf("About to free device memory, device_results\n");
  sleep(2);
  printf("Attempting to free device_results at location %p\n",(void *)device_results);
  printf("Indexed, device_results is %d",*device_results);
  CUDA_CALL(hipFree(device_results));
  printf("Freed device results\n");
  sleep(1);
  CUDA_CALL(hipFree(device_rand_state));
  printf("Freed device_rand_state\n");
  sleep(1);
  CUDA_CALL(hipFree(device_seeds));
  printf("Freed device_seeds\n");
  return 0;
}
